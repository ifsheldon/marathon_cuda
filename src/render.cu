#include "hip/hip_runtime.h"
//
// Created by Maple on 2020/12/4.
//

#include "render.cuh"
#include <iostream>

using namespace glm;

__constant__ Light Lights[MAX_LIGHT_NUM];
__constant__ Object Objects[MAX_OBJ_NUM];
__constant__ Material Materials[MAX_MATERIAL_NUM];

struct Ray
{
    vec3 origin;
    vec3 direction;
};

__constant__ float EPSILON = 0.001;
__constant__ int MAX_MARCHING_STEPS = 255;

__device__ int light_num;
__device__ int obj_num;

__device__ float sdSphere(vec3 ref_pos, float s)
{
    return length(ref_pos) - s;
}

__device__ float sdCylinder(vec3 p, float r, float h)
{
    vec2 d = abs(vec2(length(vec2(p.x, p.z)), p.y)) - vec2(r, h);
    return min(max(d.x, d.y), 0.0) + length(max(d, vec2(0.0)));
}

__device__ float calcDist(vec3 ref_point, uint objIdx, float far)
{
    vec4 refP = vec4(ref_point, 1.0);
    refP = Objects[objIdx].transformation * refP;
    ref_point = vec3(refP) / refP.w;
    switch (Objects[objIdx].shape)
    {
        case Shape::Sphere:
            return sdSphere(ref_point, Objects[objIdx].dims.x);
        case Shape::Cylinder:
            return sdCylinder(ref_point, Objects[objIdx].dims.x, Objects[objIdx].dims.y);
        default :
            return far;
    }
}

__device__ float unionSDF(float* distances, int* objIdx)
{
    float min_dist = distances[0];
    *objIdx = 0;
    for (int i = 0; i < obj_num; i++)
    {
        if (distances[i] < min_dist)
        {
            min_dist = distances[i];
            *objIdx = i;
        }
    }
    return min_dist;
}

__device__ void sceneSDF(vec3 ref_point, float* distances, float far)
{
    for (uint i = 0; i < obj_num; i++)
    {
        distances[i] = calcDist(ref_point, i, far);
    }
}

__device__ float
shortestDistanceToSurface(vec3 eye, vec3 marchingDirection, float start_dist,
                          float limit_dist,
                          int preObj,
                          float far,
                          int* objectIndex)
{
    float depth = start_dist;
    float distances[MAX_OBJ_NUM] = {0.f};
    for (int i = 0; i < MAX_MARCHING_STEPS; i++)
    {
        sceneSDF(eye + depth * marchingDirection, distances, far);
        if (preObj != -1)
            distances[preObj] = 2.0f * far;
        int hitObjIdx;
        float dist = unionSDF(distances, &hitObjIdx);
        if (dist < EPSILON)
        {
            *objectIndex = hitObjIdx;
            return depth;
        }
        depth += dist;
        if (depth >= limit_dist)
        {
            *objectIndex = obj_num;
            return limit_dist;
        }
    }
    return limit_dist;
}

__device__ vec3 estimateNormal(vec3 ref_pos, uint obj)
{
    vec4 refP = vec4(ref_pos, 1.0);
    refP = Objects[obj].transformation * refP;
    vec3 ref_point = vec3(refP) / refP.w;
    if (Objects[obj].shape == Shape::Sphere)
    {
        vec3 normal_dir = normalize(
                vec3((Objects[obj].normal_transformation * vec4(normalize(ref_point), 0.0))));
        return normal_dir;
    } else
    {
        vec2 cylinder_r_h = Objects[obj].dims;
        if (ref_point.y < 0.0)
        {
            if (abs(length(vec2(ref_point.x, ref_point.z)) - cylinder_r_h.x) >= EPSILON)
                return normalize(vec3(Objects[obj].normal_transformation * vec4(0.0, -1.0, 0.0, 0.0)));
            else
                return normalize(
                        vec3(Objects[obj].normal_transformation * vec4(ref_point.x, 0.0, ref_point.z, 0.0)));
        } else
        {
            if (abs(length(vec2(ref_point.x, ref_point.z)) - cylinder_r_h.x) >= EPSILON)
                return normalize(vec3(Objects[obj].normal_transformation * vec4(0.0, 1.0, 0.0, 0.0)));
            else
                return normalize(
                        vec3(Objects[obj].normal_transformation * vec4(ref_point.x, 0.0, ref_point.z, 0.0)));
        }
    }
}

__device__ vec3 PhongLighting(vec3 L, vec3 N, vec3 V, bool inShadow,
                              uint materialID, int lightIdx)
{
    if (inShadow)
    {
        return Lights[lightIdx].ambient * Materials[materialID].ambient;
    } else
    {
        vec3 R = reflect(-L, N);
        float N_dot_L = max(0.0, dot(N, L));
        float R_dot_V = max(0.0, dot(R, V));
        float R_dot_V_pow_n = (R_dot_V == 0.0) ? 0.0 : pow(R_dot_V, Materials[materialID].specular);
        return Lights[lightIdx].ambient * Materials[materialID].ambient +
               Lights[lightIdx].source *
               (Materials[materialID].diffuse * N_dot_L + Materials[materialID].reflect * R_dot_V_pow_n);
    }
}

__device__ vec3
castRay(const Ray* ray, const int preObj,
        const float near,
        const float far,
        const vec3 &background_color,
        bool* hasHit,
        vec3* hitPos,
        vec3* hitNormal,
        vec3* reflectDecay, int* hitObj)
{
    int objIndex;
    float dist = shortestDistanceToSurface(ray->origin, ray->direction, near, far, preObj, far, &objIndex);
    if (dist > far - EPSILON)
    {
        *hasHit = false;
        return background_color;
    } else
    {
        *hitObj = objIndex;
        *hasHit = true;
        vec3 ref_pos = ray->origin + dist * ray->direction;
        *hitPos = ref_pos;
        *hitNormal = estimateNormal(ref_pos, *hitObj);
        *reflectDecay = Materials[Objects[objIndex].material_id].reflect_decay;
        vec3 localColor = vec3(0.0);
        // shadow ray
        for (int lightIdx = 0; lightIdx < light_num; lightIdx++)
        {
            vec3 shadowRay = Lights[lightIdx].position - (*hitPos);
            Ray sRay = {*hitPos, normalize(shadowRay)};
            float max_dist = far;
            int hitObjIndex;
            float distTemp = shortestDistanceToSurface(sRay.origin, sRay.direction, EPSILON, max_dist, objIndex,
                                                       far,
                                                       &hitObjIndex);
            bool hitSth = (distTemp < max_dist - EPSILON);
            localColor += PhongLighting(sRay.direction, *hitNormal, -ray->direction, hitSth,
                                        Objects[objIndex].material_id,
                                        lightIdx);
        }
        return localColor;
    }
}

__device__ vec3 shade(const Ray* ray, const float near, const float far,
                      const int ray_marching_level, const vec3 &background_color)
{
    Ray nextRay = {ray->origin, ray->direction};
    vec3 colorResult = vec3(0.0);
    vec3 compoundedGlobalReflectDecayCoef = vec3(1.0);
    int preObj = -1;
    for (int i = 0; i < ray_marching_level; i++)
    {
        bool hasHit = false;
        vec3 hitPos, hitNormal, reflectDecay;
        vec3 localColor = castRay(&nextRay, preObj, near, far, background_color,
                                  &hasHit, &hitPos, &hitNormal, &reflectDecay, &preObj);
        colorResult += compoundedGlobalReflectDecayCoef * localColor;
        if (!hasHit)
            break;
        compoundedGlobalReflectDecayCoef *= reflectDecay;
        nextRay.origin = hitPos;
        nextRay.direction = normalize(reflect(nextRay.direction, hitNormal));
    }
    return colorResult;
}

__global__ void
renderer(const Camera camera, const CameraConfig cameraConfig, const vec2 window_size,
         float z,
         const unsigned int lightNum,
         const unsigned int objNum,
         const unsigned int ray_marching_level,
         const vec3 background_color,
         const unsigned int super_sample_rate,
         color_u8* output_colors)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int width = (int) window_size.x;
    int height = (int) window_size.y;
    if (x >= width || y >= height)
        return;

    float near = cameraConfig.config.x;
    float far = cameraConfig.config.y;
    light_num = lightNum;
    obj_num = objNum;
    float x_off = -window_size.x / 2.0f + 0.5f;
    float y_off = -window_size.y / 2.0f + 0.5f;
    z = -z;
    vec2 coord_sc = vec2(x + x_off, y + y_off);
    vec3 rayDir_ec = normalize(vec3(coord_sc, z));
    vec3 rayDir_wc = vec3(camera.look_at_mat * vec4(rayDir_ec, 0.0));
    Ray primary = {camera.position, rayDir_wc};
    vec3 colorResult_f = shade(&primary, near, far, ray_marching_level, background_color);

    float grid_size = 1.0f / super_sample_rate;
    float halt_grid_size = grid_size / 2.0f;
    float grid_base_x = x;
    float grid_base_y;
    for (unsigned int grid_x = 0; grid_x < super_sample_rate; grid_x++, grid_base_x += grid_size)
    {
        grid_base_y = y;
        for (unsigned int grid_y = 0; grid_y < super_sample_rate; grid_y++, grid_base_y += grid_size)
        {
            float rand_x = grid_base_x + halt_grid_size;
            float rand_y = grid_base_y + halt_grid_size;
            vec3 rand_ray_dir_ec = normalize(
                    vec3(rand_x + x_off, rand_y + y_off, z));
            vec3 rand_ray_dir_wc = vec3(camera.look_at_mat * vec4(rand_ray_dir_ec, 0.0));
            primary.direction = rand_ray_dir_wc;
            vec3 color_f = shade(&primary, near, far, ray_marching_level, background_color);
            colorResult_f += color_f;
        }
    }
    colorResult_f /= (super_sample_rate * super_sample_rate + 1);
    vec3 colorResult = max(min(colorResult_f * 255.f, vec3(255.f)), vec3(0.f)); // convert to [0-255]
    color_u8 color(0);
    color.r = (unsigned char) colorResult.r;
    color.g = (unsigned char) colorResult.g;
    color.b = (unsigned char) colorResult.b;
    output_colors[y * width + x] = color;
}