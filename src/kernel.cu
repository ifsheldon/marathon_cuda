#include "hip/hip_runtime.h"
//
// Created by Maple on 2020/12/4.
//

#include "kernel.cuh"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

struct Ray
{
    vec3 origin;
    vec3 direction;
};

struct Scene_d
{
    const Light* __restrict__ lights;
    const Material* __restrict__ materials;
    const Object* __restrict__ objects;
    const int light_num;
    const int obj_num;
};

#include "glm/gtc/matrix_transform.hpp"

using glm::vec4;

mat4 lookAt(vec3 eye, vec3 center, vec3 up)
{
    // Based on gluLookAt man page
    vec3 f = normalize(center - eye);
    vec3 s = normalize(cross(f, up));
    vec3 u = cross(s, f);
    return mat4(
            vec4(s, 0.0),
            vec4(u, 0.0),
            vec4(-f, 0.0),
            vec4(0.0, 0.0, 0.0, 1)
    );
}

Scene setupScene()
{
    Scene scene;
    mat4 identity(1.0);
    scene.addMaterial(vec3(1.0), vec3(0.5), vec3(1.0), vec3(0.5), 64.0f);
    scene.addMaterial(vec3(0.16, 0.14, 0.02), vec3(0.8, 0.7, 0.1), vec3(1.0), vec3(0.5), 64.0f);
    scene.addLight(vec3(0.4, -3, 0.1), vec3(0.1), vec3(1.0));
    scene.addSphere(0.5, 0, identity);
    mat4 cylinderTransformation = glm::translate(identity, vec3(-2.0, 0.0, 0.0));
    scene.addCylinder(0.5, 1.0, 0, cylinderTransformation);
    return scene;
}

void test()
{
    int size = 32;
    Scene scene = setupScene();
    dim3 dimGrid(1);
    dim3 dimBlock(size, size);
    auto mem_size = sizeof(vec3) * size * size;
    vec3* output_d;
    hipMalloc(&output_d, mem_size);
    Camera camera = {vec3(0.0, 0.0, -6.0), vec3(0.0, 1.0, 0.0), vec3(0.0),
                     lookAt(vec3(0.0, 0.0, -6.0), vec3(0.0), vec3(0.0, 1.0, 0.0))};
    CameraConfig cameraConfig = {vec3(0.01, 100.0, glm::radians(90.0))};
    float z = size / tan(cameraConfig.config.z / 2.0);
    Light* lights_d;
    hipMalloc(&lights_d, sizeof(Light) * scene.getLightNum());
    hipMemcpy(lights_d, &scene.lights[0], scene.getLightNum(), hipMemcpyHostToDevice);

    Material* materials_d;
    hipMalloc(&materials_d, sizeof(Material) * scene.getMaterialNum());
    hipMemcpy(materials_d, &scene.materials[0], scene.getMaterialNum(), hipMemcpyHostToDevice);

    Object* objects_d;
    hipMalloc(&objects_d, sizeof(Object) * scene.getObjNum());
    hipMemcpy(objects_d, &scene.objects[0], scene.getObjNum(), hipMemcpyHostToDevice);

    renderer <<< dimGrid, dimBlock>>>(1, camera, cameraConfig, vec2(size, size), z, lights_d, scene.getLightNum(),
                                      materials_d, objects_d, scene.getObjNum(),
    output_d);
    vec3* output_h = new vec3[size * size];
    hipMemcpy(output_h, output_d, mem_size, hipMemcpyDeviceToHost);
    vec3 sum(0.f);
    for (int i = 0; i < size * size; i++)
    {
        sum += output_h[i];
    }
    printf("sum = %f, %f, %f\n", sum.x, sum.y, sum.z);
    delete[] output_h;
    hipFree(output_d);
    hipFree(lights_d);
    hipFree(materials_d);
    hipFree(objects_d);
}

using namespace glm;

__device__ const int ray_marching_level = 2;
__device__ const float EPSILON = 0.001;
__device__ const int MAX_MARCHING_STEPS = 255;
__device__ const int MAX_OBJ_NUM = 50;

//TODO:
__device__ const float near = 0.0;
__device__ const float far = 100.0;

__device__ float sdSphere(vec3 ref_pos, float s)
{
    return length(ref_pos) - s;
}

__device__ float sdCylinder(vec3 p, float r, float h)
{
    vec2 d = abs(vec2(length(vec2(p.x, p.z)), p.y)) - vec2(r, h);
    return min(max(d.x, d.y), 0.0) + length(max(d, vec2(0.0)));
}

__device__ float calcDist(const Scene_d* __restrict__ scene, vec3 ref_point, uint objIdx)
{
    vec4 refP = vec4(ref_point, 1.0);
    refP = scene->objects[objIdx].transformation * refP;
    ref_point = vec3(refP) / refP.w;
    switch (scene->objects[objIdx].shape)
    {
        case Shape::Sphere:
            return sdSphere(ref_point, scene->objects[objIdx].dims.x);
        case Shape::Cylinder:
            return sdCylinder(ref_point, scene->objects[objIdx].dims.x, scene->objects[objIdx].dims.y);
        default :
            return far;
    }
}

__device__ float unionSDF(const Scene_d* __restrict__ scene, float* distances, int* objIdx)
{
    float min_dist = distances[0];
    *objIdx = 0;
    for (int i = 0; i < scene->obj_num; i++)
    {
        if (distances[i] < min_dist)
        {
            min_dist = distances[i];
            *objIdx = i;
        }
    }
    return min_dist;
}

__device__ void sceneSDF(const Scene_d* __restrict__ scene, vec3 ref_point, float* distances)
{
    for (uint i = 0; i < scene->obj_num; i++)
    {
        distances[i] = calcDist(scene, ref_point, i);
    }
}

__device__ float
shortestDistanceToSurface(const Scene_d* __restrict__ scene, vec3 eye, vec3 marchingDirection, float start_dist,
                          float limit_dist,
                          int preObj,
                          int* objectIndex)
{
    float depth = start_dist;
    float distances[MAX_OBJ_NUM] = {0.f};
    for (int i = 0; i < MAX_MARCHING_STEPS; i++)
    {
        sceneSDF(scene, eye + depth * marchingDirection, distances);
        if (preObj != -1)
            distances[preObj] = 2.0f * far;
        int hitObjIdx;
        float dist = unionSDF(scene, distances, &hitObjIdx);
        if (dist < EPSILON)
        {
            *objectIndex = hitObjIdx;
            return depth;
        }
        depth += dist;
        if (depth >= limit_dist)
        {
            *objectIndex = scene->obj_num;
            return limit_dist;
        }
    }
    return limit_dist;
}

__device__ vec3 estimateNormal(const Scene_d* __restrict__ scene, vec3 ref_pos, uint obj)
{
    vec4 refP = vec4(ref_pos, 1.0);
    refP = scene->objects[obj].transformation * refP;
    vec3 ref_point = vec3(refP) / refP.w;
    if (scene->objects[obj].shape == Shape::Sphere)
    {
        vec3 normal_dir = normalize(
                vec3((scene->objects[obj].normal_transformation * vec4(normalize(ref_point), 0.0))));
        return normal_dir;
    } else
    {
        vec2 cylinder_r_h = scene->objects[obj].dims;
        if (ref_point.y < 0.0)
        {
            if (abs(length(vec2(ref_point.x, ref_point.z)) - cylinder_r_h.x) >= EPSILON)
                return normalize(vec3(scene->objects[obj].normal_transformation * vec4(0.0, -1.0, 0.0, 0.0)));
            else
                return normalize(
                        vec3(scene->objects[obj].normal_transformation * vec4(ref_point.x, 0.0, ref_point.z, 0.0)));
        } else
        {
            if (abs(length(vec2(ref_point.x, ref_point.z)) - cylinder_r_h.x) >= EPSILON)
                return normalize(vec3(scene->objects[obj].normal_transformation * vec4(0.0, 1.0, 0.0, 0.0)));
            else
                return normalize(
                        vec3(scene->objects[obj].normal_transformation * vec4(ref_point.x, 0.0, ref_point.z, 0.0)));
        }
    }
}

__device__ vec3 PhongLighting(const Scene_d* __restrict__ scene, vec3 L, vec3 N, vec3 V, bool inShadow,
                              uint materialID, int lightIdx)
{
    if (inShadow)
    {
        return scene->lights[lightIdx].ambient * scene->materials[materialID].ambient;
    } else
    {
        vec3 R = reflect(-L, N);
        float N_dot_L = max(0.0, dot(N, L));
        float R_dot_V = max(0.0, dot(R, V));
        float R_dot_V_pow_n = (R_dot_V == 0.0) ? 0.0 : pow(R_dot_V, scene->materials[materialID].specular);
        return scene->lights[lightIdx].ambient * scene->materials[materialID].ambient +
               scene->lights[lightIdx].source *
               (scene->materials[materialID].diffuse * N_dot_L + scene->materials[materialID].reflect * R_dot_V_pow_n);
    }
}

__device__ vec3
castRay(const Ray* ray, const Scene_d* __restrict__ scene, const int preObj, bool* hasHit, vec3* hitPos,
        vec3* hitNormal,
        vec3* reflectDecay, int* hitObj)
{
    int objIndex;
    float dist = shortestDistanceToSurface(scene, ray->origin, ray->direction, near, far, preObj, &objIndex);
    if (dist > far - EPSILON)
    {
        *hasHit = false;
        return vec3(0.5); // TODO
    } else
    {
        *hitObj = objIndex;
        *hasHit = true;
        vec3 ref_pos = ray->origin + dist * ray->direction;
        *hitPos = ref_pos;
        *hitNormal = estimateNormal(scene, ref_pos, *hitObj);
        *reflectDecay = scene->materials[scene->objects[objIndex].material_id].reflect_decay;
        vec3 localColor = vec3(0.0);
        // shadow ray
        for (int lightIdx = 0; lightIdx < scene->light_num; lightIdx++)
        {
            vec3 shadowRay = scene->lights[lightIdx].position - (*hitPos);
            Ray sRay = {*hitPos, normalize(shadowRay)};
            float max_dist = far;
            int hitObjIndex;
            float distTemp = shortestDistanceToSurface(scene, sRay.origin, sRay.direction, EPSILON, max_dist, objIndex,
                                                       &hitObjIndex);
            bool hitSth = (distTemp < max_dist - EPSILON);
            localColor += PhongLighting(scene, sRay.direction, *hitNormal, -ray->direction, hitSth,
                                        scene->objects[objIndex].material_id,
                                        lightIdx);
        }
        return localColor;
    }
}

__device__ vec3 shade(const Ray* ray, const Scene_d* __restrict__ scene)
{
    Ray nextRay = {ray->origin, ray->direction};
    vec3 colorResult = vec3(0.0);
    vec3 compoundedGlobalReflectDecayCoef = vec3(1.0);
    int preObj = -1;
    for (int i = 0; i < ray_marching_level; i++)
    {
        bool hasHit = false;
        vec3 hitPos, hitNormal, reflectDecay;
        vec3 localColor = castRay(&nextRay, scene, preObj, &hasHit, &hitPos, &hitNormal, &reflectDecay, &preObj);
        colorResult += compoundedGlobalReflectDecayCoef * localColor;
        if (!hasHit)
            break;
        compoundedGlobalReflectDecayCoef *= reflectDecay;
        nextRay.origin = hitPos;
        nextRay.direction = normalize(reflect(nextRay.direction, hitNormal));
    }
    return colorResult;
}

__global__ void
renderer(const unsigned int random_seed, const Camera camera, const CameraConfig cameraConfig, const vec2 window_size,
         const float z,
         const Light* __restrict__ lights,
         const int light_num,
         const Material* __restrict__ materials,
         const Object* __restrict__ objects,
         const int obj_num,
         vec3* output_colors)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int wx = (int) window_size.x;
    int wy = (int) window_size.y;
    if (x >= wx || y >= wy)
    {
        return;
    }
    Scene_d scene = {lights, materials, objects, light_num, obj_num};
    vec2 coord_sc = vec2(x - window_size.x / 2.0f + 0.5f, y - window_size.y / 2.0f + 0.5f);
    vec3 rayDir_ec = normalize(vec3(coord_sc, -z));
    vec3 rayDir_wc = normalize(vec3(camera.look_at_mat * vec4(rayDir_ec, 0.0)));
    Ray primary = {camera.position, rayDir_wc};
    vec3 colorResult = shade(&primary, &scene);

    output_colors[y * wx + x] = vec3(1.0f);
}