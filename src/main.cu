#include <iostream>
#include <hip/hip_runtime.h>
#include "CImg.h"
#include "glm/glm.hpp"
#include "glm/gtc/matrix_transform.hpp"
#include "scene.hpp"
#include "render.cuh"

using namespace std;

static int max_x_threads = 0;
static int max_share_mem = 0;
static int warp_size = 1;

bool queryGPUCapabilitiesCUDA()
{
    // Device Count
    int devCount;

    // Get the Device Count
    hipGetDeviceCount(&devCount);

    // Print Device Count
    printf("Device(s): %i\n", devCount);
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, 0);
    std::cout << "Max Block Dim in a Grid (" << properties.maxGridSize[0] << ", " << properties.maxGridSize[1] << ", "
              <<
              properties.maxGridSize[2] << ")" << std::endl;
    std::cout << "Max Thread per Block: " << properties.maxThreadsPerBlock << std::endl;
    std::cout << "Max Thread Dim in a Block (" << (max_x_threads = properties.maxThreadsDim[0]) << ", " << properties.
            maxThreadsDim[1] <<
              ", " << properties.maxThreadsDim[2] << ")" << std::endl;
    std::cout << "Share Memory per Block (bytes): " << (max_share_mem = properties.sharedMemPerBlock) << std::endl;
    std::cout << "Mem Pitch (bytes) " << properties.memPitch << std::endl;
    std::cout << "Total Constant Memory (bytes): " << properties.totalConstMem << std::endl;
    std::cout << "Warp Size: " << (warp_size = properties.warpSize) << std::endl;
    return devCount > 0;
}


int main()
{
    if (!queryGPUCapabilitiesCUDA())
        exit(EXIT_FAILURE);
    run();
    return 0;
}
